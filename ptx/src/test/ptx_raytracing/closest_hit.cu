#include "hip/hip_runtime.h"
#include <optix.h>

using namespace optix;

rtBuffer<float4> output;
rtDeclareVariable(rtCallableProgramId<float3()>, eval, , );

RT_PROGRAM void closest_hit()
{
    float3 result = eval();
    output[0] = make_float4(result.x, result.y, result.y, 0.0);
}