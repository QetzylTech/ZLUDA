#include "hip/hip_runtime.h"
// nvcc alloca_bug.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(rtCallableProgramId<float3(float3 &mat, float3 &prd)>, sysBRDFEval, , );
rtBuffer<float3> sysMaterialParameters;

RT_PROGRAM void closest_hit()
{
	float3 mat = sysMaterialParameters[0];

	if (mat.x != 0)
	{
		const float3 texColor = make_float3(0, 0,0);
		mat = make_float3(powf(texColor.x, 2.2f), 0,0);
	}
	float3 prd2;
	float3 f = sysBRDFEval(mat, prd2);

	if (prd2.x > 0.0f)
		prd2 *= f; 
}
