#include "hip/hip_runtime.h"
// nvcc barycentrics.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtBuffer<float2, 1>   output_buffer1;
rtBuffer<float2, 1>   output_buffer2;
rtBuffer<int, 1>   output_buffer3;
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(float4,  sphere, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(int2, launch_index, rtLaunchIndex, );

RT_PROGRAM void start() {
    Ray ray = make_Ray(make_float3(float(launch_index.x), 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float  l = 1 / length(ray.direction);
  float3 D = ray.direction * l;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    float root11 = 0.0f;

    bool check_second = true;
    if( rtPotentialIntersection( (root1 + root11) * l ) ) {
      barycentrics = make_float2(100, 200);
      if(rtReportIntersection(0))
        check_second = false;
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 * l ) ) {
        barycentrics = make_float2(100, 200);
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

RT_PROGRAM void attribute_program() {
    float2 read_barycentrics = rtGetTriangleBarycentrics();
    barycentrics.x = read_barycentrics.x;
    barycentrics.y = 0.1;
}

RT_PROGRAM void closest_hit() {
    output_buffer1[launch_index.x] = barycentrics;
    // rtGetTriangleBarycentrics() happens to work here,
    // but is only valid in attribute programs
    //output_buffer2[launch_index.x] = rtGetTriangleBarycentrics();
    output_buffer3[launch_index.x] = rtGetPrimitiveIndex();
}