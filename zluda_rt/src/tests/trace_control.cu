#include "hip/hip_runtime.h"
// nvcc trace_control.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, sphere, , );
rtBuffer<uint, 1> temp_buffer;
rtBuffer<uint, 1> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, closest_distance, rtIntersectionDistance, );
rtDeclareVariable(uint, increment, attribute increment, );
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void start(void)
{
  Ray ray = make_Ray(make_float3(0, 0, -1), make_float3(0, 0, 1), 0, 0.0, RT_DEFAULT_MAX);
  char unused = 0;
  rtTrace(bvh, ray, unused);
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float l = 1 / length(ray.direction);
  float3 D = ray.direction * l;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O) - radius * radius;
  float disc = b * b - c;
  if (disc > 0.0f)
  {
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    float root11 = 0.0f;

    bool check_second = true;
    if (rtPotentialIntersection(((root1 + root11) * l) - temp_buffer[launch_index.x]))
    {
      increment = temp_buffer[launch_index.x];
      if (rtReportIntersection(0))
        check_second = false;
    }
    if (check_second)
    {
      float root2 = (-b + sdisc);
      if (rtPotentialIntersection((root2 * l) - temp_buffer[launch_index.x]))
      {
        increment = temp_buffer[launch_index.x];
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds(int, float result[6])
{
  const float3 cen = make_float3(sphere);
  const float3 rad = make_float3(sphere.w);

  optix::Aabb *aabb = (optix::Aabb *)result;

  if (rad.x > 0.0f && !isinf(rad.x))
  {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  }
  else
  {
    aabb->invalidate();
  }
}

RT_PROGRAM void any_hit_ignore(void)
{
  temp_buffer[launch_index.x] += 1;
  if (temp_buffer[launch_index.x] >= 3)
  {
    rtIgnoreIntersection();
  }
}

RT_PROGRAM void any_hit_terminate(void)
{
  temp_buffer[launch_index.x] += 1;
  if (temp_buffer[launch_index.x] >= 3)
  {
    rtTerminateRay();
  }
}

RT_PROGRAM void attribute1(void)
{
  increment = 0xc4bb2187 + temp_buffer[0];
}

RT_PROGRAM void any_hit_plus_one(void)
{
  temp_buffer[0] += 1;
}

RT_PROGRAM void attribute2(void)
{
  increment = 0xc4bb2187;
}

RT_PROGRAM void any_hit_always_ignore(void)
{
  temp_buffer[0] += 1;
  rtIgnoreIntersection();
}

RT_PROGRAM void closest_hit(void)
{
  output_buffer[launch_index.x] = increment;
}