#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4,  sphere, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, b_index, , );
rtBuffer<unsigned int, 1>   output_buffer;
rtBuffer<unsigned int, 1>   output_buffer2;
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void start(void)
{
    Ray ray = make_Ray(make_float3(0, 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void set_buffer(void)
{
    atomicAdd(&output_buffer[b_index], 1);
}

RT_PROGRAM void set_buffer2(void)
{
    atomicAdd(&output_buffer2[b_index], 1);
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float  l = 1 / length(ray.direction);
  float3 D = ray.direction * l;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    float root11 = 0.0f;

    bool check_second = true;
    if( rtPotentialIntersection( (root1 + root11) * l ) ) {
      //shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
      if(rtReportIntersection(launch_index.x))
        check_second = false;
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 * l ) ) {
        //shading_normal = geometric_normal = (O + root2*D)/radius;
        rtReportIntersection(launch_index.x);
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}
