#include "hip/hip_runtime.h"
// nvcc exception.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtBuffer<unsigned int, 1> var_buffer;
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
typedef rtCallableProgramId<unsigned int(unsigned int)> int_operator;
rtDeclareVariable(int_operator, callable1,,);
rtDeclareVariable(int_operator, callable2,,);
rtDeclareVariable(int_operator, callable3,,);

RT_PROGRAM void trace() {
    Ray ray = make_Ray(make_float3(float(launch_index.x), 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void throw_() {
    rtThrow(RT_EXCEPTION_USER);
}

RT_PROGRAM void exception() {
    var_buffer[0] = rtGetExceptionCode();
}

RT_PROGRAM void call_callable1() {
    callable1(1);
}

RT_CALLABLE_PROGRAM  unsigned int call_callable2(unsigned int x) {
    return callable2(x);
}

RT_CALLABLE_PROGRAM  unsigned int throw_callable(unsigned int x) {
    rtThrow(RT_EXCEPTION_USER + x);
    return x;
}

__noinline__ __device__ void throw_callable_sub() {
    callable3(1);
}

RT_CALLABLE_PROGRAM  unsigned int throw_callable_main(unsigned int x) {
    throw_callable_sub();
    return x;
}
