#include "hip/hip_runtime.h"
// nvcc exception.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtBuffer<unsigned int, 1> var_buffer;
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

__device__ __noinline__ void trace() {
    Ray ray = make_Ray(make_float3(float(launch_index.x), 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void start() {
    trace();
}

RT_PROGRAM void throw_() {
    rtThrow(RT_EXCEPTION_USER);
}

RT_PROGRAM void exception() {
    var_buffer[0] = rtGetExceptionCode();
}
