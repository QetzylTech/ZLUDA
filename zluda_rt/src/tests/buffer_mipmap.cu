#include "hip/hip_runtime.h"
// nvcc buffer_id.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable( uint, texture_id, , );
rtBuffer<uint2, 1>   output_buffer;

RT_PROGRAM void start() {
    bool   isResident;
    uint4 val0 = rtTex2DLodLoadOrRequest<uint4>( texture_id, 0, 0, 0, isResident );
    output_buffer[0] = make_uint2(val0.x, val0.y);
    uint4 val1 = rtTex2DLodLoadOrRequest<uint4>( texture_id, 0, 0, 1000, isResident );
    output_buffer[1] = make_uint2(val1.x, val1.y);
}
