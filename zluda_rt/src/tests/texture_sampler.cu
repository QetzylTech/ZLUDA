#include "hip/hip_runtime.h"
// nvcc texture_sampler.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtBuffer<float4, 2>   output_buffer;
rtTextureSampler<float4, 2> image1;
rtDeclareVariable(unsigned int, image2, , );

RT_PROGRAM void start() {
    output_buffer[make_uint2(0,0)] = tex2D(image1, 0, 0);
    output_buffer[make_uint2(1,0)] = tex2D(image1, 1, 0);
    output_buffer[make_uint2(0,1)] = tex2D(image1, 0, 1);
    output_buffer[make_uint2(1,1)] = tex2D(image1, 1, 1);
    output_buffer[make_uint2(0,2)] = rtTex2D<float4>(image2, 0, 0);
    output_buffer[make_uint2(1,2)] = rtTex2D<float4>(image2, 1, 0);
    output_buffer[make_uint2(0,3)] = rtTex2D<float4>(image2, 0, 1);
    output_buffer[make_uint2(1,3)] = rtTex2D<float4>(image2, 1, 1);
}
