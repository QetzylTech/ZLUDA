#include "hip/hip_runtime.h"
// nvcc get_transform.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtBuffer<float, 1> object_transforms;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(float4,  sphere, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void start() {
    Ray ray = make_Ray(make_float3(float(launch_index.x), 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float  l = 1 / length(ray.direction);
  float3 D = ray.direction * l;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    float root11 = 0.0f;

    bool check_second = true;
    if( rtPotentialIntersection( (root1 + root11) * l ) ) {
      if(rtReportIntersection(0))
      {
        rtGetTransform(RT_OBJECT_TO_WORLD, &object_transforms[16*0]);
        check_second = false;
      }
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 * l ) ) {
        if(rtReportIntersection(0))
        {
          rtGetTransform(RT_OBJECT_TO_WORLD, &object_transforms[16*0]);
        }
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  // fails compilation
  //rtGetTransform(RT_OBJECT_TO_WORLD, &object_transforms[16*0]);
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

RT_PROGRAM void any_hit() {
  rtGetTransform(RT_OBJECT_TO_WORLD, &object_transforms[16*1]);
}

RT_PROGRAM void closest_hit() {
  rtGetTransform(RT_WORLD_TO_OBJECT, &object_transforms[16*2]);
}
