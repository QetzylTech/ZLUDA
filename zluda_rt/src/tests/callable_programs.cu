// nvcc callable_programs.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(unsigned int, value, , );
rtBuffer<unsigned int, 1>   output_buffer;

typedef rtCallableProgramId<unsigned int(unsigned int)> int_operator;
rtDeclareVariable(int_operator, add_fn,,);
rtDeclareVariable(int_operator, mult_fn,,);

RT_CALLABLE_PROGRAM unsigned int add_value(unsigned int input) {
    return input + value;
}

RT_CALLABLE_PROGRAM unsigned int multiply_value(unsigned int input) {
    return input * value;
}

RT_PROGRAM void start() {    
    unsigned int x = value;
    x = add_fn(x);
    x = mult_fn(x);
    output_buffer[0] = x;
}
