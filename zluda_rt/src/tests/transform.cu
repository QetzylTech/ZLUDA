#include "hip/hip_runtime.h"
// nvcc transform.cu -I"C:\dev\OptiX SDK 6.5.0\include"  -ptx -x cu -dc
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

using namespace optix;

rtBuffer<unsigned int, 1> is_triangle;
rtBuffer<rtBufferId<float3, 1>> origin;
rtBuffer<rtBufferId<float3, 1>> direction;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, bvh, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float4,  sphere, , );

RT_PROGRAM void start() {
    Ray ray = make_Ray(make_float3(float(launch_index.x), 0, -1), make_float3(0,0,1), 0, 0.0, RT_DEFAULT_MAX);
    char unused = 0;
    rtTrace(bvh, ray, unused);
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float  l = 1 / length(ray.direction);
  float3 D = ray.direction * l;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    float root11 = 0.0f;

    bool check_second = true;
    if( rtPotentialIntersection( (root1 + root11) * l ) ) {
      if(rtReportIntersection(0))
      {
        origin[launch_index.x][0] = ray.origin;
        direction[launch_index.x][0] = ray.direction;
        check_second = false;
      }
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 * l ) ) {
        if(rtReportIntersection(0))
        {
            origin[launch_index.x][0] = ray.origin;
            direction[launch_index.x][0] = ray.direction;
        }
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

// attribute uses the same ray as related *hit function,
// so transformed for anyhit and untransformed for closesthit
RT_PROGRAM void attribute() {
    origin[launch_index.x][0] = ray.origin;
    direction[launch_index.x][0] = ray.direction;
}

RT_PROGRAM void any_hit() {
    origin[launch_index.x][1] = ray.origin;
    direction[launch_index.x][1] = ray.direction;
}

RT_PROGRAM void closest_hit() {
    origin[launch_index.x][2] = ray.origin;
    direction[launch_index.x][2] = ray.direction;
    is_triangle[launch_index.x] = rtIsTriangleHit();
}
